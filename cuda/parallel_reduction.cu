#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>  // For timing in C++

// Array size and threads per block
#define N 1000000000
#define THREADS_PER_BLOCK 256

// CUDA Kernel for parallel reduction (sum)
__global__ void reduceSum(int *input, int *output) {
    __shared__ int sharedData[THREADS_PER_BLOCK];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory
    sharedData[tid] = input[i];
    __syncthreads();

    // Reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedData[tid] += sharedData[tid + stride];
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

// CUDA Kernel for parallel reduction (max)
__global__ void reduceMax(int *input, int *output) {
    __shared__ int sharedData[THREADS_PER_BLOCK];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory
    sharedData[tid] = input[i];
    __syncthreads();

    // Reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedData[tid] = max(sharedData[tid], sharedData[tid + stride]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

// CUDA Kernel for parallel reduction (min)
__global__ void reduceMin(int *input, int *output) {
    __shared__ int sharedData[THREADS_PER_BLOCK];
    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Load elements into shared memory
    sharedData[tid] = input[i];
    __syncthreads();

    // Reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            sharedData[tid] = min(sharedData[tid], sharedData[tid + stride]);
        }
        __syncthreads();
    }

    // Write the result for this block to global memory
    if (tid == 0) {
        output[blockIdx.x] = sharedData[0];
    }
}

int main() {
    int *h_input, *h_output, *d_input, *d_output;
    int numBlocks = N / THREADS_PER_BLOCK;

    // Allocate host memory
    h_input = (int *)malloc(N * sizeof(int));
    h_output = (int *)malloc(numBlocks * sizeof(int));

    // Initialize input array with random values
    for (int i = 0; i < N; i++) {
        h_input[i] = rand() % 1000;  // Random values between 0 and 999
    }

    // Allocate device memory
    hipMalloc(&d_input, N * sizeof(int));
    hipMalloc(&d_output, numBlocks * sizeof(int));

    // Copy input data from host to device
    hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice);

    // Measure execution time for CUDA kernels
    auto cuda_start = std::chrono::high_resolution_clock::now();

    // Launch sum kernel
    reduceSum<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output);

    // Copy partial results back to host
    hipMemcpy(h_output, d_output, numBlocks * sizeof(int), hipMemcpyDeviceToHost);

    // Reduce final sum on the CPU
    int totalSum = 0;
    for (int i = 0; i < numBlocks; i++) {
        totalSum += h_output[i];
    }

    // Launch max kernel
    reduceMax<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output);
    hipMemcpy(h_output, d_output, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
    int maxVal = h_output[0];
    for (int i = 1; i < numBlocks; i++) {
        if (h_output[i] > maxVal) maxVal = h_output[i];
    }

    // Launch min kernel
    reduceMin<<<numBlocks, THREADS_PER_BLOCK>>>(d_input, d_output);
    hipMemcpy(h_output, d_output, numBlocks * sizeof(int), hipMemcpyDeviceToHost);
    int minVal = h_output[0];
    for (int i = 1; i < numBlocks; i++) {
        if (h_output[i] < minVal) minVal = h_output[i];
    }

    auto cuda_end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> cuda_duration = cuda_end - cuda_start;

    std::cout << "CUDA Sum: " << totalSum << std::endl;
    std::cout << "CUDA Max: " << maxVal << std::endl;
    std::cout << "CUDA Min: " << minVal << std::endl;
    std::cout << "CUDA Execution Time: " << cuda_duration.count() << " seconds" << std::endl;

    // Free memory
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
